#include "hip/hip_runtime.h"
#include "include/parallel_strided_sampling.cuh"
#include "include/partitioning.cuh"
#include "include/neighbor_search.cuh"
#include "include/feature_update.cuh"
#include "include/multi_hop.cuh"
#include "include/shared_aggregation.cuh"
#include "include/unique_aggregation.cuh"
#define point_cloud_file "../data/modelnet40_test.txt"


#define new_implement true
__global__ void writeToDeviceMemory(float *__restrict__ matrix, const int dim_1, const int dim_2); 

 
int main(int argc, char *argv[]) {
    hipSetDevice(7);

    const int n=1024, m=512, K=32;
    float r = 0.2;
    const int block_size = 10;

    const int block_num = block_size*block_size*block_size; 
    float step = max_coordinate/block_size + correction_factor;
    const int hop=1;
    const int search_size = 2*hop+1;
    const int search_total = search_size*search_size*search_size;
    
    feat_setting setting;
    setting.in_channel = 3;setting.out_channel_1 = 64;setting.out_channel_2 = 64;setting.out_channel_3 = 128;
    setting.bn1_m = 1;setting.bn1_var=0.5;setting.bn2_m = 0.2;setting.bn2_var=0.5;setting.bn3_m = 0.3;setting.bn3_var=0.5;

    

    float h_weight_1[setting.in_channel][setting.out_channel_1];
    float h_weight_2[setting.out_channel_1][setting.out_channel_2];
    float h_weight_3[setting.out_channel_2][setting.out_channel_3];
    
    
    for(int i=0;i < setting.out_channel_1;i++)
        for(int j=0;j < setting.in_channel;j++) 
            h_weight_1[j][i] = i+1;
    
    for(int i=0;i < setting.out_channel_2;i++)
        for(int j=0;j < setting.out_channel_1;j++) 
            h_weight_2[j][i] = i+1;

    for(int i=0;i < setting.out_channel_3;i++)
        for(int j=0;j < setting.out_channel_2;j++) 
            h_weight_3[j][i] = i+1;

    setting.weight_1 = (float*)h_weight_1;
    setting.weight_2 = (float*)h_weight_2;
    setting.weight_3 = (float*)h_weight_3;

    float *d_weight_1 = nullptr;
    float *d_weight_2 = nullptr;
    float *d_weight_3 = nullptr;

    hipStream_t load_stream = NULL;
    hipStreamCreate(&load_stream);

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_weight_1), sizeof(float) * setting.in_channel*setting.out_channel_1));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_weight_2), sizeof(float) * setting.out_channel_1*setting.out_channel_2));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_weight_3), sizeof(float) * setting.out_channel_2*setting.out_channel_3));

    CUDA_CHECK(hipMemcpyAsync(d_weight_1, h_weight_1, sizeof(float) * setting.in_channel*setting.out_channel_1, hipMemcpyHostToDevice, load_stream));
    CUDA_CHECK(hipMemcpyAsync(d_weight_2, h_weight_2, sizeof(float) * setting.out_channel_1*setting.out_channel_2, hipMemcpyHostToDevice, load_stream));
    CUDA_CHECK(hipMemcpyAsync(d_weight_3, h_weight_3, sizeof(float) * setting.out_channel_2*setting.out_channel_3, hipMemcpyHostToDevice, load_stream));
    
    hipStreamDestroy(load_stream);
    

    hipStream_t stream = NULL;
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));


    float h_c[n][3]; 
    float h_feat[n][setting.in_channel];
    
    int h_search_valid_length;

    read_point_cloud((float*)h_c, n,3, point_cloud_file);
    read_point_cloud((float*)h_feat, n,setting.in_channel,point_cloud_file);

    float *d_c = nullptr;
    float *d_feat = nullptr;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c), sizeof(float) * n*3));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_feat), sizeof(float) * n*3));
    
    CUDA_CHECK(hipMemcpyAsync(d_c, h_c, sizeof(float) * n*3, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_feat, h_feat, sizeof(float) * n*3, hipMemcpyHostToDevice, stream));   


    coord_offset h_xyz_offset = {X_Offset, Y_Offset, Z_Offset};

    int *d_point2group = nullptr;
    int *d_u_len = nullptr;
    int *d_u_offset = nullptr;
    int *d_u_order = nullptr;

//preset Level 0 Index

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_point2group), sizeof(int) * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_u_len), sizeof(int) * block_num));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_u_offset), sizeof(int) * block_num));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_u_order), sizeof(int) * n));

    
    int *d_center = nullptr;
    float *d_out_coord = nullptr;
    int h_center[m];
    float *d_temp;
    int b = 16;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_temp), sizeof(float) * n * b ));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_center), sizeof(int) * m));        
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_out_coord), sizeof(float) * m*3));

    
    int *d_search_array = nullptr;
    int *d_search_length = nullptr;
    int *d_search_offset = nullptr;
    int *d_search_valid_length = nullptr;
    int *d_len_per_group = nullptr;

//preset Level 1 Index 

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_search_array), sizeof(int) * block_num*search_total));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_search_length), sizeof(int) * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_search_offset), sizeof(int) * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_search_valid_length), sizeof(int) *1));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_len_per_group), sizeof(int) * block_num));


    float *d_mlp_result = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_mlp_result), sizeof(float) * n*setting.out_channel_3));
 
    bool *d_isn_shared = nullptr;
    bool *d_have_center = nullptr;
    int *d_1center_in_group = nullptr;
    int *d_shared_count = nullptr;
    int *d_neighbor_len = nullptr;

    //preset Level 2 Index

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_isn_shared), sizeof(bool) * block_num*n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_have_center), sizeof(bool) * block_num));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_1center_in_group), sizeof(int) * block_num));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_shared_count), sizeof(int) * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_neighbor_len), sizeof(int) * m));
 
    float *d_gather_result = nullptr;
    
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_gather_result), sizeof(float) * block_num*setting.out_channel_3));

    float *d_out_points = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_out_points), sizeof(float) * m*setting.out_channel_3));

    hipblasHandle_t cublasH = NULL;
    hipStream_t feature_stream = NULL;
 

    int *d_ns_index = nullptr;
    float *d_ns_distance = nullptr;
    int h_point2group[n];        
    int h_search_length[m];
    int h_len_per_group[block_num];
    int h_search_array[block_num*search_total];
    int h_search_offset[m];
    float *d_result1 = nullptr;
    float *d_result2 = nullptr;

    // start process
 
    partition_kernel_launcher(n, step, block_num, block_size, h_xyz_offset, d_c, d_point2group, d_u_len, d_u_offset, d_u_order, stream);

    CUDA_CHECK(hipMemcpyAsync(h_point2group, d_point2group, sizeof(int) * n, hipMemcpyDeviceToHost, stream));

    parallel_strided_sampling_kernel_wrapper(b, n, m, d_c, d_u_order ,d_center,d_out_coord, stream);

    CUDA_CHECK(hipMemcpyAsync(h_center, d_center, sizeof(int) * 1*m, hipMemcpyDeviceToHost, stream));
    
    searching_array_kernel_launcher(block_size, block_num, hop, search_size, search_total, m, d_center, d_point2group, d_u_len, d_search_array, d_search_length, d_search_offset, d_len_per_group,d_search_valid_length, stream);

    CUDA_CHECK(hipMemcpyAsync(&h_len_per_group, d_len_per_group, sizeof(int) * block_num, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_search_length, d_search_length, sizeof(int) * m, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_search_array, d_search_array, sizeof(int) * block_num*search_total, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_search_offset, d_search_offset, sizeof(int) * m, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_search_valid_length, d_search_valid_length, sizeof(int) * 1, hipMemcpyDeviceToHost, stream));
    hipDeviceSynchronize();

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_result1), sizeof(float) * n * setting.out_channel_1));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_result2), sizeof(float) * n * setting.out_channel_2));
    CUDA_CHECK(hipStreamCreateWithFlags(&feature_stream, hipStreamNonBlocking));


    CUBLAS_CHECK(hipblasCreate(&cublasH));    
    CUBLAS_CHECK(hipblasSetStream(cublasH, feature_stream));


    feature_updata(n, setting, d_feat, d_mlp_result, d_weight_1, d_weight_2, d_weight_3,cublasH, feature_stream);


    float *result = (float *)malloc(n * setting.out_channel_3 * sizeof(float));
    CUDA_CHECK(hipMemcpyAsync(result, d_mlp_result, sizeof(float) * setting.out_channel_3*n, hipMemcpyDeviceToHost,feature_stream));


 
    bool *h_isn_shared = (bool *)malloc(sizeof(bool) * block_num * n);
    int h_ns_index[h_search_valid_length];

    bool h_have_center[block_num];
    int h_1center_in_group[block_num];
    int h_shared_count[m];
    int h_neighbor_len[m];

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_ns_index), sizeof(int) * h_search_valid_length));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_ns_distance), sizeof(float) * h_search_valid_length));

    neighbor_search_kernel_launcher(n, m, block_num, K, r*r, d_c, d_center, d_point2group, d_u_len, d_u_offset, d_u_order, d_isn_shared, d_ns_index,d_ns_distance, search_total,d_search_array, d_search_length, d_search_offset, d_len_per_group, d_have_center, d_1center_in_group, d_shared_count, d_neighbor_len,stream);

    CUDA_CHECK(hipMemcpyAsync(h_ns_index, d_ns_index, sizeof(int) * h_search_valid_length, hipMemcpyDeviceToHost, stream));
 
    CUDA_CHECK(hipMemcpyAsync(h_isn_shared, d_isn_shared, sizeof(bool) * block_num*n, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_have_center, d_have_center, sizeof(bool) * block_num, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_1center_in_group, d_1center_in_group, sizeof(int) * block_num, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_shared_count, d_shared_count, sizeof(int) * m, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_neighbor_len, d_neighbor_len, sizeof(int) * m, hipMemcpyDeviceToHost, stream));

    
    shared_aggregation_kernel_launcher(n, block_num, setting.out_channel_3, K, d_center, d_have_center, d_isn_shared, d_mlp_result, d_search_length, d_search_offset, d_1center_in_group, d_ns_index, d_shared_count, d_gather_result, stream);

    unique_aggregation_kernel_launcher(n, m, setting.out_channel_3, K, d_center, d_point2group, d_mlp_result, d_ns_index, d_isn_shared, d_search_length, d_search_offset, d_gather_result, d_shared_count, d_out_points, stream);

}
 